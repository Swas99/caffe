#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <hipblas.h>

#include "caffe/layers/wino_layer.hpp"

namespace caffe {


    // dim3 threadsPerBlock(C)
    // dim3 numBlocks(Batch, nH, nW)

    // I = (Batch, H, W, C)
    // O = (16, Batch, nH, nW, C)
    template <typename T>
    __global__ void Winograd2x2ImTransCompute(const T *Input, T *Output, int C, int B, int H, int W, int pad_h, int pad_w)
    { 
        int bx = blockIdx.x; // w
        int by = blockIdx.y; // h
        int bz = blockIdx.z; // b 
        int t = threadIdx.x; // c

        int nW = (W + 1 + 2 * pad_w - 4) / 2 + 1;
        int nH = (H + 1 + 2 * pad_h - 4) / 2 + 1;

        int f_b = bz;
        int xBase = 2 * bx - pad_w;
        int yBase = 2 * by - pad_h;

        // T input_patch_1 [16] = {0};
        T input_patch_0;
        T input_patch_1;
        T input_patch_2;
        T input_patch_3;
        T input_patch_4;
        T input_patch_5;
        T input_patch_6;
        T input_patch_7;
        T input_patch_8;
        T input_patch_9;
        T input_patch_10;
        T input_patch_11;
        T input_patch_12;
        T input_patch_13;
        T input_patch_14;
        T input_patch_15;

        // load (4, 4, 1) patch of input from global memory
        int f_x, f_y;
        f_x = xBase + 0; f_y = yBase + 0;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_0 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_0 = 0;
        f_x = xBase + 1; f_y = yBase + 0;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_1 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_1 = 0;
        f_x = xBase + 2; f_y = yBase + 0;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_2 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_2 = 0;
        f_x = xBase + 3; f_y = yBase + 0;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_3 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_3 = 0;
        f_x = xBase + 0; f_y = yBase + 1;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_4 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_4 = 0;
        f_x = xBase + 1; f_y = yBase + 1;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_5 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_5 = 0;
        f_x = xBase + 2; f_y = yBase + 1;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_6 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_6 = 0;
        f_x = xBase + 3; f_y = yBase + 1;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_7 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_7 = 0;
        f_x = xBase + 0; f_y = yBase + 2;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_8 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_8 = 0;
        f_x = xBase + 1; f_y = yBase + 2;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_9 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_9 = 0;
        f_x = xBase + 2; f_y = yBase + 2;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_10 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_10 = 0;
        f_x = xBase + 3; f_y = yBase + 2;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_11 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_11 = 0;
        f_x = xBase + 0; f_y = yBase + 3;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_12 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_12 = 0;
        f_x = xBase + 1; f_y = yBase + 3;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_13 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_13 = 0;
        f_x = xBase + 2; f_y = yBase + 3;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_14 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_14 = 0;
        f_x = xBase + 3; f_y = yBase + 3;
        if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) input_patch_15 = Input [ f_b * H * W * C + f_y * W * C + f_x * C + t ]; 
        else input_patch_15 = 0;
        
        T trans_input_patch_0;
        T trans_input_patch_1;
        T trans_input_patch_2;
        T trans_input_patch_3;
        T trans_input_patch_4;
        T trans_input_patch_5;
        T trans_input_patch_6;
        T trans_input_patch_7;
        T trans_input_patch_8;
        T trans_input_patch_9;
        T trans_input_patch_10;
        T trans_input_patch_11;
        T trans_input_patch_12;
        T trans_input_patch_13;
        T trans_input_patch_14;
        T trans_input_patch_15;

        // Winograd Transform
        trans_input_patch_0 = input_patch_0 - input_patch_2 - input_patch_8 + input_patch_10;
        trans_input_patch_1 = input_patch_1 + input_patch_2 - input_patch_9 - input_patch_10;
        trans_input_patch_2 = input_patch_2 - input_patch_1 + input_patch_9 - input_patch_10;
        trans_input_patch_3 = input_patch_1 - input_patch_3 - input_patch_9 + input_patch_11;
        trans_input_patch_4 = input_patch_4 - input_patch_6 + input_patch_8 - input_patch_10;
        trans_input_patch_5 = input_patch_5 + input_patch_6 + input_patch_9 + input_patch_10;
        trans_input_patch_6 = input_patch_6 - input_patch_5 - input_patch_9 + input_patch_10;
        trans_input_patch_7 = input_patch_5 - input_patch_7 + input_patch_9 - input_patch_11;
        trans_input_patch_8 = input_patch_6 - input_patch_4 + input_patch_8 - input_patch_10;
        trans_input_patch_9 = input_patch_9 - input_patch_6 - input_patch_5 + input_patch_10;
        trans_input_patch_10 = input_patch_5 - input_patch_6 - input_patch_9 + input_patch_10;
        trans_input_patch_11 = input_patch_7 - input_patch_5 + input_patch_9 - input_patch_11;
        trans_input_patch_12 = input_patch_4 - input_patch_6 - input_patch_12 + input_patch_14;
        trans_input_patch_13 = input_patch_5 + input_patch_6 - input_patch_13 - input_patch_14;
        trans_input_patch_14 = input_patch_6 - input_patch_5 + input_patch_13 - input_patch_14;
        trans_input_patch_15 = input_patch_5 - input_patch_7 - input_patch_13 + input_patch_15;


        int offset = f_b * nH * nW * C + (by * nW + bx) * C + t;
        int stride = B * nH * nW * C;
        
        Output [ 0 * stride + offset ] = trans_input_patch_0;
        Output [ 1 * stride + offset ] = trans_input_patch_1;
        Output [ 2 * stride + offset ] = trans_input_patch_2;
        Output [ 3 * stride + offset ] = trans_input_patch_3;
        Output [ 4 * stride + offset ] = trans_input_patch_4;
        Output [ 5 * stride + offset ] = trans_input_patch_5;
        Output [ 6 * stride + offset ] = trans_input_patch_6;
        Output [ 7 * stride + offset ] = trans_input_patch_7;
        Output [ 8 * stride + offset ] = trans_input_patch_8;
        Output [ 9 * stride + offset ] = trans_input_patch_9;
        Output [ 10* stride + offset ] = trans_input_patch_10;
        Output [ 11* stride + offset ] = trans_input_patch_11;
        Output [ 12* stride + offset ] = trans_input_patch_12;
        Output [ 13* stride + offset ] = trans_input_patch_13;
        Output [ 14* stride + offset ] = trans_input_patch_14;
        Output [ 15* stride + offset ] = trans_input_patch_15;
    } 

    void Winograd2x2ImTransComputeLauncher(const Dtype *Input, float *TransIm, int C, int B, int H, int W, int pad_h, int pad_w) {
        int n_patch_width = (W + 1 + 2 * pad_w - 4) / 2 + 1;
        int n_patch_height = (H + 1 + 2 * pad_h - 4) / 2 + 1;
        dim3 blockDim(C, 1, 1);
        dim3 gridDim(n_patch_width, n_patch_height, B);
        Winograd2x2ImTransCompute<float><<<gridDim, blockDim>>>(Input, TransIm, C, B, H, W, pad_h, pad_w);
    }


// dim3 threadsPerBlock(C)
// dim3 numBlocks(Batch, nH, nW)

// Product = (16, Batch, nH, nW, K)
// Output = (Batch, H, W, K)
template <typename T>
__global__ void Output_transform(const T *Product, T *Output, int C, int B, int nH, int nW, int K, int pad_h, int pad_w)
{
    int bx = blockIdx.x; // w
    int by = blockIdx.y; // h
    int bz = blockIdx.z; // b 
    int tx = threadIdx.x; // K
    int H = 2 * nH;
    int W = 2 * nW;

    T product_patch_0 = Product [0 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_1 = Product [1 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_2 = Product [2 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_3 = Product [3 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_4 = Product [4 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_5 = Product [5 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_6 = Product [6 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_7 = Product [7 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_8 = Product [8 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_9 = Product [9 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_10= Product [10* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_11= Product [11* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_12= Product [12* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_13= Product [13* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_14= Product [14* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_15= Product [15* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    
    T output_patch_0 =  product_patch_0 + product_patch_1 + product_patch_2 + product_patch_4 +
                        product_patch_5 + product_patch_6 + product_patch_8 + product_patch_9 + product_patch_10;
    T output_patch_1 = product_patch_1 - product_patch_2 - product_patch_3 + product_patch_5 -
                       product_patch_6 - product_patch_7 + product_patch_9 - product_patch_10 - product_patch_11;
    T output_patch_2 = product_patch_4 + product_patch_5 + product_patch_6 - product_patch_8 -
                       product_patch_9 - product_patch_10 - product_patch_12 - product_patch_13 - product_patch_14;
    T output_patch_3 = product_patch_5 - product_patch_6 - product_patch_7 - product_patch_9 +
                       product_patch_10 + product_patch_11 - product_patch_13 + product_patch_14 + product_patch_15;
    
    Output[bz*H*W*K + (2*by+0)*W*K + (2*bx+0)*K + tx] = output_patch_0;
    Output[bz*H*W*K + (2*by+0)*W*K + (2*bx+1)*K + tx] = output_patch_1;
    Output[bz*H*W*K + (2*by+1)*W*K + (2*bx+0)*K + tx] = output_patch_2;
    Output[bz*H*W*K + (2*by+1)*W*K + (2*bx+1)*K + tx] = output_patch_3;
} 

__global__ void assign(const float *Input, const float *Weight, float *tmp_data_buffer, const float **Input_ptrs_gpu, const float **Weight_ptrs_gpu, float **tmp_product_ptrs_gpu, int C, int B, int nH, int nW, int K) {
    int tx = threadIdx.x; // 16
    
    Input_ptrs_gpu[tx] = Input + tx * B * nH * nW * C;
    Weight_ptrs_gpu[tx] = Weight + tx * K * C;
    tmp_product_ptrs_gpu[tx] = tmp_data_buffer + tx * nH * nW * B * K;
}

// Input = (16, B, nH, nW, C)
// Weight = (16, C, K)
void Winograd2x2ConvComputeLauncher(const float *Input, const float *Weight, float *Output, float *tmp_data_buffer, const long long *tmp_ptr_buffer, int C, int B, int nH, int nW, int K, int pad_h, int pad_w) {

    const float** Input_ptrs_gpu_ = (const float **)(tmp_ptr_buffer);
    const float** Weight_ptrs_gpu_ = (const float **)(tmp_ptr_buffer + 16);
    float** tmp_product_ptrs_gpu_ = (float **)(tmp_ptr_buffer + 16 * 2);

    dim3 bDim(16, 1, 1);
    dim3 gDim(1, 1, 1);
    assign <<<gDim, bDim>>> (Input, Weight, tmp_data_buffer, Input_ptrs_gpu_, Weight_ptrs_gpu_, tmp_product_ptrs_gpu_, C, B, nH, nW, K);
    
    float one = 1;
    float zero = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, B * nH * nW, C,
        &one,
        Weight_ptrs_gpu_, K,
        Input_ptrs_gpu_, C,
        &zero, tmp_product_ptrs_gpu_, K, 16);

    dim3 blockDim2(K, 1, 1);
    dim3 gridDim2(nW, nH, B);
    Output_transform <float> <<<gridDim2, blockDim2>>> (tmp_data_buffer, Output, C, B, nH, nW, K, pad_h, pad_w);

    hipblasDestroy(handle);
}


    void xxx(const float *input, const float *weights, float *output, int B,int H,int W,int pad_h,int pad_w, int C, int K) {
         
        // kernel_dim_; 

        int nW = (W + 1) / 2;
        int nH = (H + 1) / 2;
        float *wTransInput;
        hipMalloc((void **)&wTransInput, 16* B* nH * nW * C* sizeof(float));
        hipMemset(wTransInput,0, 16* B* nH * nW * C* sizeof(float));
        
        Winograd2x2ImTransComputeLauncher(input, wTransInput, C, B, H, W,1,1);



        float *Output;
        hipMalloc((void **)&Output, B* 2*nH * 2*nW * K * sizeof(float));
        hipMemset(Output,0, B* 2*nH * 2*nW * K * sizeof(float));    

        // Allocate temporary memory
        float *tmp_data_buffer_tensor;
        hipMalloc((void **)&tmp_data_buffer_tensor, 16 * nH * nW * B * K * sizeof(float));
        
        long long *tmp_ptr_buffer_tensor;
        hipMalloc((void **)&tmp_ptr_buffer_tensor, 3 * 16 * sizeof(long long));


        // Set all but the first element of the output tensor to 0.
         Winograd2x2ConvComputeLauncher(wTransInput, weights, output, 
         tmp_data_buffer_tensor, tmp_ptr_buffer_tensor, C, B, nH, nW, K, 1, 1); 

        hipFree(wTransInput);
        hipFree(tmp_ptr_buffer_tensor);
        hipFree(tmp_data_buffer_tensor);
    
    }


    void WinogradLayer<Dtype>::compute_output_shape() {
        const int *kernel_shape_data = this->kernel_shape_.gpu_data();
        const int *stride_data = this->stride_.gpu_data();
        const int *pad_data = this->pad_.gpu_data();
        const int *dilation_data = this->dilation_.gpu_data();
        this->output_shape_.clear();
        for (int i = 0; i < this->num_spatial_axes_; ++i) {
            // i + 1 to skip channel axis
            const int input_dim = this->input_shape(i + 1);
            const int kernel_extent = dilation_data[i] * (kernel_shape_data[i] - 1) + 1;
            const int output_dim = (input_dim + 2 * pad_data[i] - kernel_extent)
                                   / stride_data[i] + 1;
            this->output_shape_.push_back(output_dim);
        }
    }

    template<typename Dtype>
    void WinogradLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                              const vector<Blob<Dtype> *> &top) {
        const Dtype *weight = this->blobs_[0]->gpu_data();
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype *bottom_data = bottom[i]->gpu_data();
            Dtype *top_data = top[i]->mutable_gpu_data();


            int H,W,pad_h,pad_w,C;
            this->get_input_height(H);
            this->get_input_width(W);
            this->get_pad_height(pad_h);
            this->get_pad_width(pad_w);
            this->get_conv_in_channels(C);

            const int *kernel_shape_data = this->kernel_shape_.gpu_data();

            //printf("B: %d \n", this->num_);
            //printf("C: %d \n", C);
            //printf("input_h: %d \n", H);
            //printf("input_w: %d \n", W);
            //printf("pad_h: %d \n", pad_h);
            //printf("pad_w: %d \n", pad_w);
            printf("K: %d \n", kernel_shape_data[0]);
            printf("K2: %d \n", kernel_shape_data[1]);
            printf("K3: %d \n", kernel_shape_data[2]);
            printf("K4: %d \n", kernel_shape_data[3]);
            printf("K5: %d \n", kernel_shape_data[4]);
            printf("K6: %d \n", kernel_shape_data[9]);
            printf("K7: %d \n", kernel_shape_data[18]);
            xxx(bottom_data, weight, top_data, this->num_,H,W,pad_h,pad_w,C,kernel_shape_data[0]);

            //for (int n = 0; n < this->num_; ++n) {
            //    if (kernel_shape_data[i] < 3) //kernel size !=3 has not implemented
            //        this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
            //                               top_data + n * this->top_dim_);
            //    else {
            //        //this->forward_gpu_winograd(bottom_data + n * this->bottom_dim_, weight,
            //        //                           top_data + n * this->top_dim_);
            //        this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
            //                               top_data + n * this->top_dim_);
            //    }

            //    if (this->bias_term_) {
            //        const Dtype *bias = this->blobs_[1]->gpu_data();
            //        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
            //    }
            //}
        }
    }

    template<typename Dtype>
    void WinogradLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
                                               const vector<bool> &propagate_down,
                                               const vector<Blob<Dtype> *> &bottom) {
        const Dtype *weight = this->blobs_[0]->gpu_data();
        Dtype *weight_diff = this->blobs_[0]->mutable_gpu_diff();
        for (int i = 0; i < top.size(); ++i) {
            const Dtype *top_diff = top[i]->gpu_diff();
            const Dtype *bottom_data = bottom[i]->gpu_data();
            Dtype *bottom_diff = bottom[i]->mutable_gpu_diff();
            // Bias gradient, if necessary.
            if (this->bias_term_ && this->param_propagate_down_[1]) {
                Dtype *bias_diff = this->blobs_[1]->mutable_gpu_diff();
                for (int n = 0; n < this->num_; ++n) {
                    this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
                }
            }
            if (this->param_propagate_down_[0] || propagate_down[i]) {
                for (int n = 0; n < this->num_; ++n) {
                    // gradient w.r.t. weight. Note that we will accumulate diffs.
                    if (this->param_propagate_down_[0]) {
                        this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
                                              top_diff + n * this->top_dim_, weight_diff);
                    }
                    // gradient w.r.t. bottom data, if necessary.
                    if (propagate_down[i]) {
                        this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
                                                bottom_diff + n * this->bottom_dim_);
                    }
                }
            }
        }
    }



    INSTANTIATE_LAYER_GPU_FUNCS(WinogradLayer);

}  // namespace caffe
