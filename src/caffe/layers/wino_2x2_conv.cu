#include "hip/hip_runtime.h"
#include "caffe/blob.hpp"
#include "caffe/layer.hpp"
#include "caffe/proto/caffe.pb.h"

#include "caffe/layers/base_wino_2x2.hpp"

namespace caffe {


// dim3 threadsPerBlock(C)
// dim3 numBlocks(Batch, nH, nW)

// Product = (16, Batch, nH, nW, K)
// Output = (Batch, H, W, K)
template <typename T>
__global__ void Output_transform(const T *Product, T *Output, int C, int B, int nH, int nW, int K, int pad_h, int pad_w)
{
    int bx = blockIdx.x; // w
    int by = blockIdx.y; // h
    int bz = blockIdx.z; // b 
    int tx = threadIdx.x; // K
    int H = 2 * nH;
    int W = 2 * nW;
    
    T product_patch_0 = Product [0 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_1 = Product [1 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_2 = Product [2 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_3 = Product [3 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_4 = Product [4 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_5 = Product [5 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_6 = Product [6 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_7 = Product [7 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_8 = Product [8 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_9 = Product [9 * B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_10= Product [10* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_11= Product [11* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_12= Product [12* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_13= Product [13* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_14= Product [14* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    T product_patch_15= Product [15* B * nH * nW * K + bz * nH * nW * K + by * nW * K + bx * K + tx];
    
    T output_patch_0 =  product_patch_0 + product_patch_1 + product_patch_2 + product_patch_4 +
                        product_patch_5 + product_patch_6 + product_patch_8 + product_patch_9 + product_patch_10;
    T output_patch_1 = product_patch_1 - product_patch_2 - product_patch_3 + product_patch_5 -
                       product_patch_6 - product_patch_7 + product_patch_9 - product_patch_10 - product_patch_11;
    T output_patch_2 = product_patch_4 + product_patch_5 + product_patch_6 - product_patch_8 -
                       product_patch_9 - product_patch_10 - product_patch_12 - product_patch_13 - product_patch_14;
    T output_patch_3 = product_patch_5 - product_patch_6 - product_patch_7 - product_patch_9 +
                       product_patch_10 + product_patch_11 - product_patch_13 + product_patch_14 + product_patch_15;
    
    Output[bz*H*W*K + (2*by+0)*W*K + (2*bx+0)*K + tx] = output_patch_0;
    Output[bz*H*W*K + (2*by+0)*W*K + (2*bx+1)*K + tx] = output_patch_1;
    Output[bz*H*W*K + (2*by+1)*W*K + (2*bx+0)*K + tx] = output_patch_2;
    Output[bz*H*W*K + (2*by+1)*W*K + (2*bx+1)*K + tx] = output_patch_3;


    //printf("Output patch:\n");
    //printf("%.2f %.2f %.2f %.2f\n", output_patch_0,output_patch_1,output_patch_2,output_patch_3);
        
} 


__global__ void assign(const float *Input, const float *Weight, float *tmp_data_buffer, const float **Input_ptrs_gpu, const float **Weight_ptrs_gpu, float **tmp_product_ptrs_gpu, int C, int B, int nH, int nW, int K) {
    int tx = threadIdx.x; // 16
    
    Input_ptrs_gpu[tx] = Input + tx * B * nH * nW * C;
    Weight_ptrs_gpu[tx] = Weight + tx * K * C;
    tmp_product_ptrs_gpu[tx] = tmp_data_buffer + tx * nH * nW * B * K;
}

// Input = (16, B, nH, nW, C)
// Weight = (16, C, K)

void Winograd2x2ConvComputeLauncher(const float *Input, const float *Weight, float *Output, float *tmp_data_buffer, const long long *tmp_ptr_buffer, int C, int B, int nH, int nW, int K, int pad_h, int pad_w) {

    const float** Input_ptrs_gpu_ = (const float **)(tmp_ptr_buffer);
    const float** Weight_ptrs_gpu_ = (const float **)(tmp_ptr_buffer + 16);
    float** tmp_product_ptrs_gpu_ = (float **)(tmp_ptr_buffer + 16 * 2);

    dim3 bDim(16, 1, 1);
    dim3 gDim(1, 1, 1);
    assign <<<gDim, bDim>>> (Input, Weight, tmp_data_buffer, Input_ptrs_gpu_, Weight_ptrs_gpu_, tmp_product_ptrs_gpu_, C, B, nH, nW, K);
    
    float one = 1;
    float zero = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, B * nH * nW, C,
        &one,
        Weight_ptrs_gpu_, K,
        Input_ptrs_gpu_, C,
        &zero, tmp_product_ptrs_gpu_, K, 16);

    dim3 blockDim2(K, 1, 1);
    dim3 gridDim2(nW, nH, B);
    Output_transform <float> <<<gridDim2, blockDim2>>> (tmp_data_buffer, Output, C, B, nH, nW, K, pad_h, pad_w);

    hipblasDestroy(handle);
}



    // void xxx(const float *input, const float *weights, float *output, int B,int H,int W,int pad_h,int pad_w, int C, int K) {
         
    //     // kernel_dim_; 

    //     int nW = (W + 1) / 2;
    //     int nH = (H + 1) / 2;
    //     float *wTransInput;
    //     hipMalloc((void **)&wTransInput, 16* B* nH * nW * C* sizeof(float));
    //     hipMemset(wTransInput,0, 16* B* nH * nW * C* sizeof(float));
        
    //     Winograd2x2ImTransComputeLauncher(input, wTransInput, C, B, H, W,1,1);


    //     hipMalloc((void **)&output, B* 2*nH * 2*nW * K * sizeof(float));
    //     hipMemset(output,0, B* 2*nH * 2*nW * K * sizeof(float));    

    //     // Allocate temporary memory
    //     float *tmp_data_buffer_tensor;
    //     hipMalloc((void **)&tmp_data_buffer_tensor, 16 * nH * nW * B * K * sizeof(float));
        
    //     long long *tmp_ptr_buffer_tensor;
    //     hipMalloc((void **)&tmp_ptr_buffer_tensor, 3 * 16 * sizeof(long long));


    //     // Set all but the first element of the output tensor to 0.
    //     Winograd2x2ConvComputeLauncher(wTransInput, weights, output, 
    //     tmp_data_buffer_tensor, tmp_ptr_buffer_tensor, C, B, nH, nW, K, 1, 1); 

    //     hipFree(wTransInput);
    //     hipFree(tmp_ptr_buffer_tensor);
    //     hipFree(tmp_data_buffer_tensor);
    
    // }


    void xxx(const double *input, const double *weights, double *output, int B,int H,int W,int pad_h,int pad_w, int C, int K) {
         
    }



    template<typename Dtype>
    void Winograd2x2ConvLayer<Dtype>::compute_output_shape() {
        const int *kernel_shape_data = this->kernel_shape_.gpu_data();
        const int *stride_data = this->stride_.gpu_data();
        const int *pad_data = this->pad_.gpu_data();
        const int *dilation_data = this->dilation_.gpu_data();
        this->output_shape_.clear();
        for (int i = 0; i < this->num_spatial_axes_; ++i) {
            // i + 1 to skip channel axis
            const int input_dim = this->input_shape(i + 1);
            const int kernel_extent = dilation_data[i] * (kernel_shape_data[i] - 1) + 1;
            const int output_dim = (input_dim + 2 * pad_data[i] - kernel_extent)
                                   / stride_data[i] + 1;
            this->output_shape_.push_back(output_dim);
        }
    }

    template<typename Dtype>
    void Winograd2x2ConvLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                              const vector<Blob<Dtype> *> &top) {
        const Dtype *weight = this->blobs_[0]->gpu_data();
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype *bottom_data = bottom[i]->gpu_data();
            Dtype *top_data = top[i]->mutable_gpu_data();


            //int H,W,pad_h,pad_w,C;
            //this->get_input_height(H);
            //this->get_input_width(W);
            //this->get_pad_height(pad_h);
            //this->get_pad_width(pad_w);
            //this->get_conv_in_channels(C);
            const int *kernel_shape_data = this->kernel_shape_.cpu_data();

            //printf("B: %d \n", this->num_);
            //printf("C: %d \n", C);
            //printf("input_h: %d \n", H);
            //printf("input_w: %d \n", W);
            //printf("pad_h: %d \n", pad_h);
            //printf("pad_w: %d \n", pad_w);
            //printf("K: %d \n", kernel_shape_data[i]);
            //xxx(bottom_data, weight, top_data, this->num_,H,W,pad_h,pad_w,C,kernel_shape_data[i]);

            for (int n = 0; n < this->num_; ++n) {
                //printf("K: %d \n", kernel_shape_data[i]);
                if (kernel_shape_data[i] < 3) //kernel size !=3 has not implemented
                    this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
                                           top_data + n * this->top_dim_);
                else {
                    //this->forward_gpu_winograd(bottom_data + n * this->bottom_dim_, weight,
                    //                           top_data + n * this->top_dim_);
                    this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
                                           top_data + n * this->top_dim_);
                }
                if (this->bias_term_) {
                    const Dtype *bias = this->blobs_[1]->gpu_data();
                    this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
                }
            }
        }
    }

    
    template<typename Dtype>
    void Winograd2x2ConvLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
                                               const vector<bool> &propagate_down,
                                               const vector<Blob<Dtype> *> &bottom) {

        const Dtype *weight = this->blobs_[0]->gpu_data();
        Dtype *weight_diff = this->blobs_[0]->mutable_gpu_diff();
        for (int i = 0; i < top.size(); ++i) {
            const Dtype *top_diff = top[i]->gpu_diff();
            const Dtype *bottom_data = bottom[i]->gpu_data();
            Dtype *bottom_diff = bottom[i]->mutable_gpu_diff();
            // Bias gradient, if necessary.
            if (this->bias_term_ && this->param_propagate_down_[1]) {
                Dtype *bias_diff = this->blobs_[1]->mutable_gpu_diff();
                for (int n = 0; n < this->num_; ++n) {
                    this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
                }
            }
            if (this->param_propagate_down_[0] || propagate_down[i]) {
                for (int n = 0; n < this->num_; ++n) {
                    // gradient w.r.t. weight. Note that we will accumulate diffs.
                    if (this->param_propagate_down_[0]) {
                        this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
                                              top_diff + n * this->top_dim_, weight_diff);

                    }
                    // gradient w.r.t. bottom data, if necessary.
                    if (propagate_down[i]) {
                        this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
                                                bottom_diff + n * this->bottom_dim_);
                        //this->forward_gpu_gemm(top_diff + n * this->top_dim_, weight,
                        //                        bottom_diff + n * this->bottom_dim_);
                    }
                }
            }
        }
    }

    INSTANTIATE_LAYER_GPU_FUNCS(Winograd2x2ConvLayer);

}  // namespace caffe